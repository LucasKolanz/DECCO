#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "../cuVectorMath.h"
#include <iostream>

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

hipError_t addWithCuda(double3* pos, const double3* vel, const double3* acc, unsigned int size);

__global__ void addKernel(double3* pos, const double3* vel, const double3* acc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	pos[i] = vel[i] * acc[i];
}

int main()
{
	const int arraySize = 5;
	double3* pos = new double3[arraySize];
	double3* vel = new double3[arraySize];
	double3* acc = new double3[arraySize];

	for (size_t i = 0; i < arraySize; i++)
	{
		vel[i] = make_double3((double)i, (double)i, (double)i);
		acc[i] = make_double3((double)i, (double)i, (double)i);
	}

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(pos, vel, acc, arraySize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (size_t i = 0; i < arraySize; i++)
	{
		printf("\nHost data:\n", vel[i].x, vel[i].y, vel[i].z);
		printf("Velocity:\t%lf\t%lf\t%lf\n", vel[i].x, vel[i].y, vel[i].z);
		printf("Accel:   \t%lf\t%lf\t%lf\n", acc[i].x, acc[i].y, acc[i].z);
		printf("Position:\t%lf\t%lf\t%lf\n", pos[i].x, pos[i].y, pos[i].z);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double3* pos, const double3* vel, const double3* acc, unsigned int size)
{

	double3* dev_pos;
	double3* dev_vel;
	double3* dev_acc;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_pos, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_vel, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_acc, size * sizeof(double3));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_vel, vel, size * sizeof(double3), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_acc, acc, size * sizeof(double3), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.

	dim3 numBlocks(1, 1, 1);
	dim3 tpb(size, 1, 1);

	addKernel << <numBlocks, tpb >> > (dev_pos, dev_vel, dev_acc);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(pos, dev_pos, size * sizeof(double3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_pos);
	hipFree(dev_vel);
	hipFree(dev_acc);

	return cudaStatus;
}
