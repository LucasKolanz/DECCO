#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "../cuVectorMath.h"
#include <iostream>
#include <omp.h>
#include "../misc.h"


hipError_t double3Math(double3* pos, const double3* vel, const double3* acc, unsigned int size);

__global__ void addKernel(double3* pos, const double3* vel, const double3* acc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	pos[i] = vel[i] * acc[i];
}



int main()
{
	// Data size
	int const fullSet = 1 << 15;
	int pairs = fullSet * (fullSet - 1) / 2;
	int2* pair = new int2[pairs];
	int count = 0;

	// Create random ball positions.
	double3* pos = new double3[fullSet];
	for (size_t i = 0; i < fullSet; i++)
	{
		pos[i] = make_double3(randDouble(1000), randDouble(1000), randDouble(1000));
	}
	double* dist = new double[pairs];

	// Create unique pairs.
	for (size_t i = 0; i < fullSet; i++)
	{
		for (size_t j = i + 1; j < fullSet; j++)
		{
			pair[count] = make_int2(i, j);
			count++;
		}
	}

	printf("Combinations = %d\n", pairs);


	double time0 = 0;
	double time1 = 0;

	/////////////////////////////////////////
	// Run pair strategy
	time0 = omp_get_wtime();
	for (size_t i = 0; i < pairs; i++)
	{
		dist[i] = mag(pos[pair[i].x] - pos[pair[i].y]);
		//printf("%d\t%d\n", pair[i].x, pair[i].y);
	}
	time1 = omp_get_wtime();
	printf("Time = %lf\n", time1 - time0);

	//// Print for optimization
	double sum = 0;
	for (size_t i = 0; i < fullSet; i++)
	{
		sum += dist[i];
	}

	printf("sum = %lf\n", sum);

	/////////////////////////////////////////
	// Run double loop with counter to minimize dist array size.
	count = 0;
	time0 = omp_get_wtime();
	for (size_t i = 0; i < fullSet; i++)
	{
		for (size_t j = i + 1; j < fullSet; j++)
		{
			dist[count] = mag(pos[i] - pos[j]);
			count++;
		}
	}
	time1 = omp_get_wtime();
	printf("Time = %lf\n", time1 - time0);

	//// Print for optimization
	sum = 0;
	for (size_t i = 0; i < fullSet; i++)
	{
		sum += dist[i];
	}

	printf("sum = %lf\n", sum);

	/////////////////////////////////////////
	// Without counter
	time0 = omp_get_wtime();
	for (size_t i = 0; i < fullSet; i++)
	{
		for (size_t j = i + 1; j < fullSet; j++)
		{
			dist[i] = mag(pos[i] - pos[j]);
		}
	}
	time1 = omp_get_wtime();
	printf("Time = %lf\n", time1 - time0);

	//// Print for optimization
	sum = 0;
	for (size_t i = 0; i < fullSet; i++)
	{
		sum += dist[i];
	}
	printf("sum = %lf\n", sum);


	//const int arraySize = 5;
	//double3* pos = new double3[arraySize];
	//double3* vel = new double3[arraySize];
	//double3* acc = new double3[arraySize];

	//for (size_t i = 0; i < arraySize; i++)
	//{
	//	vel[i] = { (double)i, (double)i, (double)i };
	//	acc[i] = { (double)i,(double)i, (double)i };
	//}

	//// Add vectors in parallel.
	//hipError_t cudaStatus = double3Math(pos, vel, acc, arraySize);
	//if (cudaStatus != hipSuccess)
	//{
	//	fprintf(stderr, "addWithCuda failed!");
	//	return 1;
	//}

	//for (size_t i = 0; i < arraySize; i++)
	//{
	//	printf("\nHost data:\n", vel[i].x, vel[i].y, vel[i].z);
	//	printf("Velocity:\t%lf\t%lf\t%lf\n", vel[i].x, vel[i].y, vel[i].z);
	//	printf("Accel:   \t%lf\t%lf\t%lf\n", acc[i].x, acc[i].y, acc[i].z);
	//	printf("Position:\t%lf\t%lf\t%lf\n", pos[i].x, pos[i].y, pos[i].z);
	//}


	//// hipDeviceReset must be called before exiting in order for profiling and
	//// tracing tools such as Nsight and Visual Profiler to show complete traces.
	//cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess)
	//{
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	return 1;
	//}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t double3Math(double3* ans, const double3* a, const double3* b, unsigned int size)
{

	double3* dev_ans;
	double3* dev_a;
	double3* dev_b;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_ans, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double3));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double3), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double3), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	// Later set blockSize to 64 or something to make multiple warps per SM.
	dim3 numBlocks(1, 1, 1);
	dim3 tpb(size, 1, 1);

	addKernel << <numBlocks, tpb >> > (dev_ans, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess)
	//{
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	goto Error;
	//}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(ans, dev_ans, size * sizeof(double3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_ans);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
