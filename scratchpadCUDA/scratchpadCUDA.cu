#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <iostream>
#include <fstream>
#include <time.h>
#include <sstream>
#include <stdio.h>
#include "math.h"

#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include "../cuVectorMath.h"
#include "../initializations.h"
#include "../misc.h"
#include "../objects.h"


hipError_t double3Math(double3* pos, const double3* vel, const double3* acc, unsigned int size);

__global__ void addKernel(double3* pos, const double3* vel, const double3* acc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	pos[i] += vel[i] * acc[i];
}



int main()
{
	const int arraySize = 5;
	double3* pos = new double3[arraySize];
	double3* vel = new double3[arraySize];
	double3* acc = new double3[arraySize];

	for (size_t i = 0; i < arraySize; i++)
	{
		vel[i] = { (double)i, (double)i, (double)i };
		acc[i] = { (double)i,(double)i, (double)i };
	}

	// Add vectors in parallel.
	hipError_t cudaStatus = double3Math(pos, vel, acc, arraySize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	for (size_t i = 0; i < arraySize; i++)
	{
		printf("Velocity:	\t%lf\t%lf\t%lf\n", vel[i].x, vel[i].y, vel[i].z);
		printf("Accel:		\t%lf\t%lf\t%lf\n", acc[i].x, acc[i].y, acc[i].z);
		printf("Position:	\t%lf\t%lf\t%lf\n", pos[i].x, pos[i].y, pos[i].z);
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t double3Math(double3* ans, const double3* a, const double3* b, unsigned int size)
{

	double3* dev_ans;
	double3* dev_a;
	double3* dev_b;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_ans, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double3));
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double3));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double3), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double3), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	// Later set blockSize to 64 or something to make multiple warps per SM.
	dim3 numBlocks(1, 1, 1);
	dim3 tpb(size, 1, 1);

	for (size_t i = 0; i < 10; i++)
	{
		addKernel << <numBlocks, tpb >> > (dev_ans, dev_a, dev_b);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess)
	//{
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	goto Error;
	//}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(ans, dev_ans, size * sizeof(double3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_ans);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
